#include "hip/hip_runtime.h"
#ifndef _KERNELS_CU_
#define _KERNELS_CU_

#include "../headers/particles.h"
#include "../utils/cuda_vector_math.cuh"
#include "../utils/cuda_device.h"
#include "../utils/simple_io.h"

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/reduce.h>



__constant__ Movementparams par_dev;

hipError_t __host__ ParticleSystem::copyParamsToDevice(){
	return hipMemcpyToSymbol(HIP_SYMBOL(par_dev), &par, sizeof(Movementparams));
}

// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
// KERNEL to set up RANDOM GENERATOR STATES
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
__global__ void rngStateSetup_kernel(int * rng_Seeds, hiprandState * rngStates, int N){
	//int tid = threadIdx.x;							// each block produces exactly the same random numbers
	int tid_u = threadIdx.x + blockIdx.x*blockDim.x;	// each block produces different random numbers
	if (tid_u >= N) return;
	
	hiprand_init (rng_Seeds[tid_u], 0, 0, &rngStates[tid_u]);
}

#define PP_SEED 777

void ParticleSystem::initRNG(){
	srand(PP_SEED);
	for (int i=0; i<N; ++i) seeds_h[i] = rand(); 
	hipMemcpy( seeds_dev, seeds_h, sizeof(int)*N, hipMemcpyHostToDevice);

	int nt = 256, nb = (N-1)/nt+1;
	rngStateSetup_kernel <<< nb, nt>>> (seeds_dev, dev_XWstates, N);
	getLastCudaError("psys_RNG_kernel_launch");
}


// =========================================================================================
//		Interacting Paticles Kernels!!
// =========================================================================================

// given the position, get the cell ID on a square grid of dimensions nxGrid x nxGrid,
// with each cell of size cellSize
// this function returns cellId considering 0 for 1st grid cell. With multiple blocks, user must add appropriate offset
//		|---|---|---|---|---|
//		|   |   |   |   |   |
//		|---|---|---|---|---|
//		|   |   | x |   |   |	<-- x = (pos.x, pos.y)
//		|---|---|---|---|---|
//		|   |   |   |   |   |
//		|---|---|---|---|---|
//      ^ 0 = (xmin, ymin)	^ nx = xmin + nx*cellSize
inline __device__ int getCellId(float2 pos){
	int ix = (pos.x - par_dev.xmin)/(par_dev.cellSize+1e-12);	// add 1e-6 to make sure that particles on edge of last cell are included in that cell
	int iy = (pos.y - par_dev.ymin)/(par_dev.cellSize+1e-12);
	return iy*par_dev.nCellsX + ix;
}


__global__ void update_grid_kernel(float2* pos_array, int* cellCount_array, int* cellId_array, int* cellParticles_array){
	unsigned int pid = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (pid < par_dev.N){
		int ic = getCellId(pos_array[pid]);//, s);	// get cell Id ic of particle pid
		cellId_array[pid] = ic;						// store cell id in array
		int n = atomicAdd(&cellCount_array[ic],1);	// atomic increment particle count of cell ic
//		n = clamp(n,0,3);
		cellParticles_array[ic*4+n] = pid;		// add particle index to cell ic
	}
}


// CAUTION: pos_new and vel_new are never initialized. Never use them before initializing.
__global__ void movement_kernel(float2 *pos, float2* vel, float2* vel_new, float* stk,
								int * cellParticles, int * cellCounts, int * cellIds){
	
	unsigned int myId = blockIdx.x*blockDim.x + threadIdx.x;	// full particle ID
	if (myId >= par_dev.N) return;
	
	float2 dirR = make_float2(0,0);
	float2 dirA = make_float2(0,0);
//	float2 dirO = make_float2(0,0);
	
	float2 myPos = pos[myId];
	float2 myVel = vel[myId];
	float  mystk = stk[myId];
	
	int nCellsScan = int(par_dev.Rs*0.99999999)+1;
	int myCell     = cellIds[myId];
	int myCellx    = myCell % par_dev.nCellsX;		// convert grid cell to x and y indices
	int myCelly    = myCell / par_dev.nCellsX;
	
	for (int innx=-nCellsScan; innx<nCellsScan+1; ++innx){			//  offsets to add in x and y indices to get neighbour cells
		for (int inny=-nCellsScan; inny<nCellsScan+1; ++inny){
			int otherCellx = myCellx + innx;
			otherCellx = otherCellx + int(otherCellx < 0)*par_dev.nCellsX - int(otherCellx >= par_dev.nCellsX)*par_dev.nCellsX;
			int otherCelly = myCelly + inny;
			otherCelly = otherCelly + int(otherCelly < 0)*par_dev.nCellsX - int(otherCelly >= par_dev.nCellsX)*par_dev.nCellsX;
			
			int nnCell = ix2(otherCellx, otherCelly, par_dev.nCellsX);
			
			for (int ip=0; ip < cellCounts[nnCell]; ++ip){ // i is particle index within nnCell (max = 3)
				
				int i = cellParticles[nnCell*4+ip];	// particle ip in cell nncell
				//printf("thread %d, myCell %d,%d otherCell %d,%d otherParticle %d\n", myId, myCellx,myCelly, otherCellx,otherCelly,i ); // << "thread" << "\n"

				if (i == myId) continue;	// Exclude self
			
				// get direction and distance to other 
				float2 v2other = periodicDisplacement(	myPos, pos[i], 
														par_dev.xmax-par_dev.xmin, 
														par_dev.ymax-par_dev.ymin  );
				float d2other = length(v2other);
		
				// indicator variables 
				float Irr = float(d2other < par_dev.Rr); //? 1:0;
				float Ira = float(d2other < par_dev.Rs); //? 1:0;
		
				// keep adding to dirR and dirA so that average direction or R/A will be taken
				v2other = normalize(v2other); // normalise to consider direction only

				dirR = dirR - v2other*Irr;				// add repulsion only if other fish lies in inside Rr
				dirA = dirA + v2other*Ira*(1-Irr); 		// add attraction only if other fish lies in (Rr < r < Ra)
//				dirO = dirO + vel[i]*Ira*(1-Irr);	// add alignment only if other fish lies in (Rr < r < Ra)
			}
			
		}	
	}

	float Ir = float(length(dirR) > 1e-6);	// particles in Rr
	float Ia = float(length(dirA) > 1e-6);	// particles in Ra and hence also in Rr

	float2 dirS = dirA; //myVel*(1-mystk) + dirA*mystk;	

	float2 finalDir = /*myVel*(1-Ir)*(1-Ia) +*/ dirR*Ir + dirS*Ia*(1-Ir);	
	finalDir = normalizeSafeZero(finalDir);	// finalDir can be zero, as this is just dirR or dirS.
	
	vel_new[myId] = finalDir;
}

// seperate integration in different kernel to keep register usage < 63 
__global__ void integrate_kernel(float2* pos, float2* vel, float2 *dirs_new, float* stk, float2* velField, int nxt, int nyt,
								 hiprandState * RNGstates){
	unsigned int myId = blockIdx.x*blockDim.x + threadIdx.x;	// full particle ID
	if (myId >= par_dev.N) return;

	float2 myVel = vel[myId];
	float2 myPos = pos[myId];
	float2 finalDir = dirs_new[myId];

//	// apply turning rate constraint
//	float sinT = myVel.x*finalDir.y - myVel.y*finalDir.x;		// sinT = myVel x finalDir
//	float cosT = dot(finalDir, myVel);	// Desired turning angle. Both vectors are unit so dot product is cos(theta) 
//	float cosL = clamp( max(cosT, par_dev.cosphi), -1.f, 1.f);
//	float sinL = sqrtf(1-cosL*cosL);
//	sinL = sinL - 2*sinL*float(sinT < 0);	// equivalent to: if (sinT < 0) sinL = -sinL;
//	float2 a = make_float2(myVel.x*cosL - myVel.y*sinL, myVel.x*sinL + myVel.y*cosL);

	int ipx = (nxt-1)*(myPos.x-par_dev.xmin)/(par_dev.xmax-par_dev.xmin);
	int ipy = (nyt-1)*(myPos.y-par_dev.ymin)/(par_dev.ymax-par_dev.ymin);
	float2 flowVel = velField[ipy*(nxt-1)+ipx];

	// generate copy error (copy error must be zero if finalDir = 0)
	float2 eta = float(length(finalDir)>1e-6)*hiprand_normal2(&RNGstates[myId])*par_dev.copyErrSd;

	myVel = flowVel*1 + stk[myId]*(finalDir+eta)*par_dev.speed; //normalize(a)*par_dev.speed;

	float2 netVel = myVel; //*1 + flowVel*5*0;
	
	myPos = myPos + netVel * par_dev.dt; 
	makePeriodic(myPos.x, par_dev.xmin, par_dev.xmax);
	makePeriodic(myPos.y, par_dev.ymin, par_dev.ymax);

	pos[myId] = myPos; 
	vel[myId] = finalDir;  /// ??? Check with Vishu
}


// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
// Kernels and Launchers for grouping algo 
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

__global__ void distance_kernel(float2* pos, int myId, int* otherIds){
	unsigned int otherId = blockIdx.x*blockDim.x + threadIdx.x;	// full particle ID
	if (otherId >= par_dev.N) return;

	// get direction and distance to other 
	float2 v2other = periodicDisplacement(	pos[myId], pos[otherId], 
											par_dev.xmax-par_dev.xmin, 
											par_dev.ymax-par_dev.ymin  );
	float d2other = length(v2other);

	otherIds[otherId] = int(d2other < par_dev.rGrp);
}


int ParticleSystem::launch_distance_kernel(int myId){
	distance_kernel <<<gridDims, blockDims>>> (pos_dev, myId, closeIds_dev);
	return 0;
}


__device__ int sort_count;	// keeps track of the number of close particle pairs obtained until reset

void ParticleSystem::resetSortCount(){
	int atomicCount = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(sort_count), &atomicCount, sizeof(int), 0, hipMemcpyHostToDevice);
}

__global__ void atomic_sort_kernel(int myId, int* otherIds, int2* pairs){
	unsigned int otherId = blockIdx.x*blockDim.x + threadIdx.x;	// full particle ID
	if (otherId >= par_dev.N) return;
	if (otherIds[otherId] == 1) pairs[atomicAdd(&sort_count,1)] = make_int2(myId, otherId);
	//__syncthreads();
}


int ParticleSystem::launch_atomic_sort_kernel(int myId){
	int count;
	atomic_sort_kernel <<<gridDims, blockDims>>> (myId, closeIds_dev, closeParticlePairs_dev);
	hipMemcpyFromSymbol(&count, HIP_SYMBOL(sort_count), sizeof(int), 0, hipMemcpyDeviceToHost);
	return count;
}


void print_devArray(int * vdev, int n, int ncol=-1, bool row = false){
	int * v = new int[n];
	hipMemcpy(v, vdev, n*sizeof(int), hipMemcpyDeviceToHost);
	if (ncol == -1) printArray(v,n);
	else{
		cout << "particles in cell:\n";
		for (int i=0; i<n/ncol; ++i){
			if (row) cout << i << " | ";
			for (int j=0; j<ncol; ++j){
				cout << v[i*ncol+j] << " ";
			}
			cout << "\n";
		}
		cout << "\n";
	}
	delete [] v;
}


void ParticleSystem::launch_movement_kernel(){

	// reset all cell counts to 0
	thrust::fill( (thrust::device_ptr <int>)cellCounts_dev, (thrust::device_ptr <int>)cellCounts_dev + par.nCellsXY, (int)0);
	
	update_grid_kernel <<<gridDims, blockDims>>>(pos_dev, cellCounts_dev, cellIds_dev, cellParticles_dev);
	getLastCudaError("update_grid_kernel");

    movement_kernel <<<gridDims, blockDims>>>(pos_dev, vel_dev, vel_new_dev, stk_dev, 
    										  cellParticles_dev, cellCounts_dev, cellIds_dev);

	integrate_kernel <<<gridDims, blockDims>>>(pos_dev, vel_dev, vel_new_dev, stk_dev, 
											   psTE->vel_field_dev, psTE->nx, psTE->ny,
											   dev_XWstates); 

//	print_devArray(cellCounts_dev, par.nCellsXY, par.nCellsX);
//	print_devArray(cellParticles_dev, par.nCellsXY*4, 4, true);
//	print_devArray(cellIds_dev, par.N);
	
}


#endif



